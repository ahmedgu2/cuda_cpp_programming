#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <random>

#define KERNEL_SIZE 7
#define TILE_DIM 32
__constant__ float kernel[KERNEL_SIZE * KERNEL_SIZE];


#define CUDA_CHECK_ERROR(callResult) do{ \
    hipError_t error = callResult; \
    if(error != hipSuccess){ \
        std::cout << "Error in " << __FILE__ << " at line " << __LINE__ << "\n" << hipGetErrorString(error); \
        exit(EXIT_FAILURE); \
    } \
}while(0)

#define CUDA_KERNEL_CHECK_ERROR() do{ \
    hipError_t error = hipGetLastError(); \
    if(error != hipSuccess){ \
        std::cerr << "----CUDA ERROR in " << __FILE__ << " at line " << __LINE__ << "\n" << hipGetErrorString(error) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}while(0)


// Utils
void initMatrix(float *matrix, int nRows, int nCols, unsigned int seed = 42){
    std::mt19937 gen(seed);
    std::uniform_int_distribution<int> dist(0, 255);
    for(int i = 0; i < nRows; ++i){
        for(int j = 0; j < nCols; ++j){
            matrix[i * nCols + j] = dist(gen);
        }
    }
}

void printMatrix(float *matrix, int nRows, int nCols){
    for(int i = 0; i < nRows; ++i){
        for(int j = 0; j < nCols; ++j){
            std::cout << matrix[i * nRows + j] << " ";
        }
        std::cout << std::endl;
    }
}

// GPU version
float normal_pdf2D(float x, const float y, const float std){
    const float pi = 3.141592653589;
    const float coeff = 1 / (2 * pi * std * std);
    const float exponent = - 0.5 * ((x * x + y * y) / (std * std));
    return coeff * exp(exponent);
}

void initKernel2D(float *kernel, const int kernelSize){
    // Since kernelSize is odd, it's written as 2k + 1. k in here is the center of the kernel grid.
    // To apply the gaussian correctly, we need to express (i, j) relative to the center of the grid (k, k).
    const int k = (kernelSize - 1) / 2;  
    float sum = 0.f; // Normalization 
    for(int i = 0; i < kernelSize; ++i){
        for(int j = 0; j < kernelSize; ++j){
            kernel[i * kernelSize + j] = normal_pdf2D(i - k, j - k, 1.0);
            sum += kernel[i * kernelSize + j];
        }
    }
    for(int i = 0; i < kernelSize; ++i){
        for(int j = 0; j < kernelSize; ++j){
            kernel[i * kernelSize + j] /= sum;
        }
    }
}

__device__
int clamp(const int x, const int min, const int max){
   if(x < min) return min;
   if(x > max) return max;
   return x;
}

__global__ 
void convoluteOptimized_gpu(float *result, float *mat, const int nRows, const int nCols){
    // We're using edge expension padding, so dimension of result and mat are equal.
    __shared__ float tile[TILE_DIM + KERNEL_SIZE - 1][TILE_DIM + KERNEL_SIZE - 1];
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int k = (KERNEL_SIZE - 1) / 2;
    // Shared memory indices
    int shared_x = tx + k;
    int shared_y = ty + k;

    // Load tile in shared memory
    if(row < nRows && col < nCols){
        tile[shared_y][shared_x] = mat[row * nCols + col];
    }else{
        int clampedCol = clamp(col, 0, nCols - 1);
        int clampedRow = clamp(row, 0, nRows);
        tile[ty][tx] = mat[clampedRow * nCols + clampedCol];
    }
    __syncthreads();


    if(row < nRows && col < nCols){
        int indx = row * nCols + col;
        result[indx] = 0;

        for(int i = -k; i <= k; ++i){
            for(int j = -k; j <= k; ++j){
                int curRow = clamp(row + i, 0, nRows - 1);
                int curCol = clamp(col + j, 0, nCols - 1);
                result[indx] += mat[curRow * nCols + curCol] * kernel[(i + k) * KERNEL_SIZE + (j + k)];
            }
        }
    }
}

void gaussianBlurOptimized_gpu(
    float *mat,
    float *result,
    const int nRows,
    const int nCols
){
   // We consider that the KernelSize in here is at most 7x7, thus it fits in a single block.
    float *d_mat, *d_result;
    float *d_sum;
    float h_kernel[KERNEL_SIZE * KERNEL_SIZE];

    initKernel2D(h_kernel, KERNEL_SIZE);

    CUDA_CHECK_ERROR(hipMalloc(&d_result, nRows * nCols * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc(&d_mat, nRows * nCols * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc(&d_sum, sizeof(float)));

    // Init sum
    CUDA_CHECK_ERROR(hipMemset(d_sum, 0, sizeof(float)));
    CUDA_CHECK_ERROR(hipMemcpy(d_mat, mat, nRows * nCols * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(kernel), h_kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(float)));

    dim3 threadsPerBlock(TILE_DIM, TILE_DIM);
    dim3 numBlocks((nCols + threadsPerBlock.x - 1) / threadsPerBlock.x, (nRows + threadsPerBlock.y - 1) / threadsPerBlock.y);
    convoluteOptimized_gpu<<<numBlocks, threadsPerBlock>>>(d_result, d_mat, nRows, nCols);
    CUDA_KERNEL_CHECK_ERROR();

    CUDA_CHECK_ERROR(hipMemcpy(result, d_result, nRows * nCols * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_mat);
    hipFree(d_result);
    hipFree(d_sum);
}

int main(){

    int nRows = 1024, nCols = 1024;
    float *mat = new float[nRows * nCols];
    float *result = new float[nRows * nCols];

    initMatrix(mat, nRows, nCols);

    gaussianBlurOptimized_gpu(mat, result, nRows, nCols);

    delete[] mat;
    delete[] result;
}