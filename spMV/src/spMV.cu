#include "hip/hip_runtime.h"
#include "cuda_macros.cuh"
#include "COOMatrixGPU.cuh"
#include "CSRMatrixGPU.cuh"

__global__
void spmv_coo(COOMatrixGPU mat, float *x, float *y){
    int indx = threadIdx.x + blockDim.x * blockIdx.x;
    if(indx < mat.getNumNonZeros()){
        uint32_t row = mat.getRowIndx(indx);
        uint32_t col = mat.getColIndx(indx);
        float val = mat.getValue(indx);
        atomicAdd(&y[row], val * x[col]);
    }
}

__global__
void spmv_csr(CSRMatrixGPU mat, float *x, float *y){
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    if(row < mat.getnRows()){
        float sum = 0.f;
        for(size_t i = mat.getRowPtrs(row); i < mat.getRowPtrs(row + 1); ++i){
            uint32_t col = mat.getColsIndx(i);
            sum += mat.getValues(i) * x[col];
        }
        y[row] = sum;
    }
}

void spmvCoo_gpu(COOMatrixGPU& mat, size_t nRows, size_t nCols, float *x, float *y){
    float *d_x, *d_y;

    CUDA_CHECK_ERROR(hipMalloc(&d_x, nCols * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc(&d_y, nRows * sizeof(float)));

    CUDA_CHECK_ERROR(hipMemset(d_y, 0, nRows * sizeof(float)));

    CUDA_CHECK_ERROR(hipMemcpy(d_x, x, nCols * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(d_y, y, nRows * sizeof(float), hipMemcpyHostToDevice));

    size_t length = mat.getNumNonZeros();
    int BLOCK_DIM = 1024;
    int GRID_DIM = (length + BLOCK_DIM - 1) / BLOCK_DIM;
    // FIXME: Passing by value calls the desctructor after kernel finishes, which will make mat unusable afterwards.
    spmv_coo<<<GRID_DIM, BLOCK_DIM>>>(mat, d_x, d_y);
    CUDA_KERNEL_CHECK_ERROR();
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR(hipMemcpy(y, d_y, nRows * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK_ERROR(hipFree(d_x));
    CUDA_CHECK_ERROR(hipFree(d_y));
}

void spmvCSR_gpu(CSRMatrixGPU& mat, size_t nRows, size_t nCols, float *x, float *y){
    float *d_x, *d_y;

    CUDA_CHECK_ERROR(hipMalloc(&d_x, nCols * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc(&d_y, nRows * sizeof(float)));

    CUDA_CHECK_ERROR(hipMemset(d_y, 0, nRows * sizeof(float)));

    CUDA_CHECK_ERROR(hipMemcpy(d_x, x, nCols * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(d_y, y, nRows * sizeof(float), hipMemcpyHostToDevice));

    size_t length = mat.getNumNonZeros();
    int BLOCK_DIM = 1024;
    int GRID_DIM = (length + BLOCK_DIM - 1) / BLOCK_DIM;
    // FIXME: Passing by value calls the desctructor after kernel finishes, which will make mat unusable afterwards.
    spmv_csr<<<GRID_DIM, BLOCK_DIM>>>(mat, d_x, d_y);
    CUDA_KERNEL_CHECK_ERROR();
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR(hipMemcpy(y, d_y, nRows * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK_ERROR(hipFree(d_x));
    CUDA_CHECK_ERROR(hipFree(d_y));
}