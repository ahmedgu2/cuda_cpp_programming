
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>

#define CUDA_CHECK_ERROR(callResult) do{ \
    hipError_t error = callResult; \
    if(error != hipSuccess){ \
        std::cout << "Error in " << __FILE__ << " at line " << __LINE__ << "\n" << hipGetErrorString(error); \
        exit(EXIT_FAILURE); \
    } \
}while(0)

#define CUDA_KERNEL_CHECK_ERROR() do{ \
    hipError_t error = hipGetLastError(); \
    if(error != hipSuccess){ \
        std::cerr << "----CUDA ERROR in " << __FILE__ << " at line " << __LINE__ << "\n" << hipGetErrorString(error) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}while(0)


float** createMatrix(int nRows, int nCols){
    float **matrix = new float*[nRows];
    for(int i = 0; i < nRows; ++i)
        matrix[i] = new float[nCols];
    return matrix;
}

void initMatrix(float **matrix, int nRows, int nCols){
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist;
    for(int i = 0; i < nRows; ++i){
        for(int j = 0; j < nCols; ++j){
            matrix[i][j] = dist(gen);
        }
    }
}

void printMatrix(float **matrix, int nRows, int nCols){
    for(int i = 0; i < nRows; ++i){
        for(int j = 0; j < nCols; ++j){
            std::cout << matrix[i][j] << " ";
        }
        std::cout << std::endl;
    }
}

float* flattenMatrix(float **mat, int nRows, int nCols){
    float *flatMat = new float[nRows * nCols];
    for(int i = 0; i < nRows; ++i){
        for(int j = 0; j < nCols; ++j){
            flatMat[i * nCols + j] = mat[i][j];
        }
    }
    return flatMat;
}

void freeMatrix(float **mat, int nRows){
    for(int i = 0; i < nRows; ++i)
        delete[] mat[i];
    delete mat;
}

void mm_cpu(float **mat1, float **mat2, float **result, int nRows1, int nCols1, int nRows2, int nCols2){
    if(nCols1 != nRows2){
        std::cerr << "ERROR: nCols1 != nRows1. These must be equal to be able to apply matrix multiplcation!" << std::endl;
        exit(EXIT_FAILURE);
    }
    for(int i = 0; i < nRows1; ++i){
        for(int j = 0; j < nCols2; ++j){
            result[i][j] = 0.f;
            for(int k = 0; k < nCols1; ++k){
                result[i][j] += mat1[i][k] * mat2[k][j];
            }
        }
    }
}

__global__
void mmNaive_gpu(float *mat1, float *mat2, float *result, int nRows1, int nCols1, int nRows2, int nCols2){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    if(i < nRows1 && j < nCols2){
        float sum = 0.f;
        for(int k = 0; k < nCols1; ++k)
            sum += mat1[i * nCols1 + k] * mat2[k * nCols2 + j];
        result[i * nCols2 + j] = sum;
    }
}

int main(){
    int nRows1 = 512, nRows2 = 256, nCols1 = 256, nCols2 = 512;
    float **mat1 = createMatrix(nRows1, nCols1);
    float **mat2 = createMatrix(nRows2, nCols2);
    float **result = createMatrix(nRows1, nCols2);
    
    initMatrix(mat1, nRows1, nCols1);
    initMatrix(mat2, nRows2, nCols2);

    // Device vars
    float *d_mat1, *d_mat2, *d_result;
    size_t size1 = nRows1 * nCols1 * sizeof(float);
    size_t size2 = nRows2 * nCols2 * sizeof(float);
    size_t sizer = nRows1 * nCols2 * sizeof(float);

    CUDA_CHECK_ERROR(hipMalloc(&d_mat1, size1));
    CUDA_CHECK_ERROR(hipMalloc(&d_mat2, size2));
    CUDA_CHECK_ERROR(hipMalloc(&d_result, sizer));

    // Flatten host 2d arrays to copy them to device memory
    float *mat1Flat = flattenMatrix(mat1, nRows1, nCols1);
    float *mat2Flat = flattenMatrix(mat2, nRows2, nCols2);
    float *resultGPU = flattenMatrix(result, nRows1, nCols2);

    // Copy matrices to device
    CUDA_CHECK_ERROR(hipMemcpy(d_mat1, mat1Flat, size1, hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(d_mat2, mat2Flat, size2, hipMemcpyHostToDevice));

    // Create events for computing runtime
    hipEvent_t start, end;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&end));

    // Launch Kernel
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((nRows1 + threadsPerBlock.x - 1) / threadsPerBlock.x, (nCols2 + threadsPerBlock.y - 1) / threadsPerBlock.y);
    CUDA_CHECK_ERROR(hipEventRecord(start));

    mmNaive_gpu<<<numBlocks, threadsPerBlock>>>(d_mat1, d_mat2, d_result, nRows1, nCols1, nRows2, nCols2);

    CUDA_CHECK_ERROR(hipEventRecord(end));
    CUDA_CHECK_ERROR(hipEventSynchronize(end));
    CUDA_KERNEL_CHECK_ERROR();

    // Copy result to host
    CUDA_CHECK_ERROR(hipMemcpy(resultGPU, d_result, sizer, hipMemcpyDeviceToHost));

    // Kernel runtime
    float gpuTimeMilliSeconds = 0.f;
    CUDA_CHECK_ERROR(hipEventElapsedTime(&gpuTimeMilliSeconds, start, end));

    // CPU version for testing
    auto cpuStart = std::chrono::high_resolution_clock::now();
    mm_cpu(mat1, mat2, result, nRows1, nCols1, nRows2, nCols2);
    auto cpuEnd = std::chrono::high_resolution_clock::now();
    auto cpuTimeMicroseconds = std::chrono::duration_cast<std::chrono::microseconds>(cpuEnd - cpuStart).count();

    // Print timing
    std::cout << "CPU function runtime: " << cpuTimeMicroseconds << " us" << std::endl;
    std::cout << "GPU kernel runtime: " << gpuTimeMilliSeconds * 1000 << " us" << std::endl;
    std::cout << "Speedup: " << cpuTimeMicroseconds / (gpuTimeMilliSeconds * 1000) << std::endl;

    // Check output matching
    std::cout << "Comparing outputs..." << std::endl;
    bool testFailed = false;
    for(int i = 0; i < nRows1; ++i){
        for(int j = 0; j < nCols2; ++j){
            if((result[i][j] - resultGPU[i * nCols2 + j]) > 1e-4){
                std::cout << "Mismatch at index " << i << "," << j << ": Expected " << result[i][j] << ", Found " << resultGPU[i * nCols2 + j] << std::endl;
                testFailed = true;
            }
        }
    }
    if(testFailed)
        std::cout << "\033[31mTEST FAILED\033[0m" << std::endl;
    else
        std::cout << "\033[32mTEST PASSED!\033[0m" << std::endl;

    // Free memory
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_result);
    freeMatrix(mat1, nRows1);
    freeMatrix(mat2, nRows2);
    freeMatrix(result, nRows1);
    free(mat1Flat);
    free(mat2Flat);
    free(resultGPU);
    return 0;
}