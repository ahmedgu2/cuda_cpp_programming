
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>
#include <chrono>

#define cudaErrorCheck(callReturn) do{ \
    hipError_t error = callReturn; \
    if(error != hipSuccess){ \
        std::cerr << "----CUDA ERROR in " << __FILE__ << " at line " << __LINE__ << "\n" << hipGetErrorString(error) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}while(0)

#define cudaKernelErrorCheck() do{ \
    hipError_t error = hipGetLastError(); \
    if(error != hipSuccess){ \
        std::cerr << "----CUDA ERROR in " << __FILE__ << " at line " << __LINE__ << "\n" << hipGetErrorString(error) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}while(0);


std::vector<float> generateRandomVector(size_t n, float minValue, float maxValue){
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(minValue, maxValue);
    
    std::vector<float> vector(n);
    for(int i = 0; i < n; ++i){
        vector[i] = dist(gen);
    }
    return vector;
}

__global__
void vectorAddition_gpu(float *d_v1, float *d_v2, float *output, size_t n){
    // grid-stride loop
    for(int indx = threadIdx.x + blockIdx.x * blockDim.x; indx < n; indx += gridDim.x * blockDim.x){
        output[indx] = d_v1[indx] + d_v2[indx];
    }
}

std::vector<float> vectorAddition_cpu(std::vector<float>& v1, std::vector<float>& v2){
    std::vector<float> output(v1.size());
    for(int i = 0; i < v1.size(); ++i)
        output[i] = v1[i] + v2[i];
    return output;
}


int main(){
    size_t N = 1e8;
    std::vector<float> v1 = generateRandomVector(N, 0.f, 5.f);
    std::vector<float> v2 = generateRandomVector(N, 0.f, 5.f);
    std::vector<float> gpuOutput(N);
    
    float *d_v1, *d_v2, *d_output;
    size_t sizeBytes = N * sizeof(float);
    // Init device arrays
    cudaErrorCheck(hipMalloc((void**)&d_v1, sizeBytes));
    cudaErrorCheck(hipMalloc((void**)&d_v2, sizeBytes));
    cudaErrorCheck(hipMalloc((void**)&d_output, sizeBytes));

    // Copy host data to device
    cudaErrorCheck(hipMemcpy(d_v1, v1.data(), sizeBytes, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(d_v2, v2.data(), sizeBytes, hipMemcpyHostToDevice));

    // Create events for timing the kernel
    hipEvent_t startTime, endTime;
    cudaErrorCheck(hipEventCreate(&startTime));
    cudaErrorCheck(hipEventCreate(&endTime));

    // Run Kernel
    dim3 threadsPerBlock(512);
    dim3 numBlocks(1024);
    
    cudaErrorCheck(hipEventRecord(startTime));
    vectorAddition_gpu<<<numBlocks, threadsPerBlock>>>(d_v1, d_v2, d_output, N);
    cudaErrorCheck(hipEventRecord(endTime));
    cudaErrorCheck(hipEventSynchronize(endTime));
    cudaKernelErrorCheck();
    
    // Calculate GPU time
    float gpuTimeMilliSeconds = 0.f;
    cudaErrorCheck(hipEventElapsedTime(&gpuTimeMilliSeconds, startTime, endTime));

    // Copy result to host
    cudaErrorCheck(hipMemcpy(gpuOutput.data(), d_output, sizeBytes, hipMemcpyDeviceToHost));

    // CPU version for testing
    auto cpuStart = std::chrono::high_resolution_clock::now();
    auto cpuOutput = vectorAddition_cpu(v1, v2); 
    auto cpuEnd = std::chrono::high_resolution_clock::now();
    auto cpuTimeMicroseconds = std::chrono::duration_cast<std::chrono::microseconds>(cpuEnd - cpuStart).count();

    // Print timing
    std::cout << "CPU function runtime: " << cpuTimeMicroseconds << " us" << std::endl;
    std::cout << "GPU kernel runtime: " << gpuTimeMilliSeconds * 1000 << " us" << std::endl;
    std::cout << "Speedup: " << cpuTimeMicroseconds / (gpuTimeMilliSeconds * 1000) << std::endl;

    // Check output matching
    std::cout << "Comparing outputs..." << std::endl;
    bool testFailed = false;
    for(int i = 0; i < v1.size(); ++i){
        if(gpuOutput[i] != cpuOutput[i]){
            std::cout << "Mismatch at index " << i << ": Expected " << cpuOutput[i] << ", Found " << gpuOutput[i] << std::endl;
            testFailed = true;
        }
    }
    if(testFailed)
        std::cout << "\033[31mTEST FAILED\033[0m" << std::endl;
    else
        std::cout << "\033[32mTEST PASSED!\033[0m" << std::endl;
    // test result
     
    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_output);
    return 0;
}