#include "cuda_macros.cuh"

void allocateCudaMemory(float **ptr, size_t size){
    CUDA_CHECK_ERROR(hipMalloc(ptr, size * sizeof(float)));
}

void freeCudaMemory(float *ptr){
    CUDA_CHECK_ERROR(hipFree(ptr));
}

void copyToCuda(float *dst_ptr, float *src_ptr, size_t size){
    CUDA_CHECK_ERROR(hipMemcpy(dst_ptr, src_ptr, size * sizeof(float), hipMemcpyHostToDevice));
}