#include "cuda_macros.cuh"

void allocateCudaMemory(float **ptr, size_t size){
    CUDA_CHECK_ERROR(hipMalloc(&ptr, size * sizeof(float)));
}

void freeCudaMemory(float *ptr){
    CUDA_CHECK_ERROR(hipFree(ptr));
}