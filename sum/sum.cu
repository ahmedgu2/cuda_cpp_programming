
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

#define CUDA_CHECK_ERROR(callResult) do{ \
    hipError_t error = callResult; \
    if(error != hipSuccess){ \
        std::cout << "Error in " << __FILE__ << " at line " << __LINE__ << "\n" << hipGetErrorString(error); \
        exit(EXIT_FAILURE); \
    } \
}while(0)

#define CUDA_KERNEL_CHECK_ERROR() do{ \
    hipError_t error = hipGetLastError(); \
    if(error != hipSuccess){ \
        std::cerr << "----CUDA ERROR in " << __FILE__ << " at line " << __LINE__ << "\n" << hipGetErrorString(error) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}while(0)

// Utils
void initVector(float *vector, const int length, unsigned int seed = 42){
    std::mt19937 gen(seed);
    std::uniform_int_distribution<int> dist(0, 255);
    for(int i = 0; i < length; ++i){
        vector[i] = dist(gen);
    }
}

void printVector(float *vector, const int length){
    for(int i = 0; i < length; ++i){
        std::cout << vector[i] << " ";
    }
    std::cout << std::endl;
}

float sum_cpu(float *input, const int length){
    float sum_ = 0;
    for(int i = 0; i < length; ++i){
        sum_ += input[i];
    }
    return sum_;
}

void test_cpu_gpu(float sum_cpu_result, float sum_gpu_result){
    std::cout << "sum_cpu : " << sum_cpu_result << ", sum_gpu: " << sum_gpu_result << std::endl;
    if(sum_cpu_result != sum_gpu_result)
        std::cout << "\033[31mTEST FAILED\033[0m" << std::endl;
    else
        std::cout << "\033[32mTEST PASSED!\033[0m" << std::endl;    
}

__global__
void simpleSumReduction_gpu(float *input, const int length, float *output){
    /**
     * Simple sum kernel for input arrays that fit in 1 block.
     */
    const uint32_t indx = 2 * threadIdx.x;
    for(int stride = 1; stride <= blockDim.x; stride *= 2){
        if(threadIdx.x % stride == 0){
            input[indx] += input[indx + stride];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0){
        *output = input[0];
    }
}

__global__
void simpleSumReductionV2_gpu(float *input, const int length, float *output){
    /**
     * Optimizes `simpleSumReduction_gpu()` by decreasing control divergence. 
     * This is done by applying a better thread assignement strategy.
     */
    for(int stride = blockDim.x; stride >= 1; stride /= 2){
        if(threadIdx.x < stride){
            input[threadIdx.x] += input[threadIdx.x + stride];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0){
        *output = input[0];
    }
}


int main(){
    const int length = 8;
    float *input = new float[length];
    float sum_cpu_result, sum_gpu_result;
    float *d_input, *d_sum_gpu_result;
    
    // Initialize input data
    initVector(input, length);

    // Compute CPU sum
    sum_cpu_result = sum_cpu(input, length);

    // Allocate GPU memory
    CUDA_CHECK_ERROR(hipMalloc(&d_input, length * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc(&d_sum_gpu_result, sizeof(float)));

    // Copy input data to GPU
    CUDA_CHECK_ERROR(hipMemcpy(d_input, input, length * sizeof(float), hipMemcpyHostToDevice));

    // Launch kernel
    // Since we're using threadIdx.x * 2 in the kernel, we need length/2 threads
    int numThreads = length / 2;
    simpleSumReductionV2_gpu<<<1, numThreads>>>(d_input, length, d_sum_gpu_result);
    CUDA_KERNEL_CHECK_ERROR();

    // Copy result back to host
    CUDA_CHECK_ERROR(hipMemcpy(&sum_gpu_result, d_sum_gpu_result, sizeof(float), hipMemcpyDeviceToHost));

    // Compare results
    test_cpu_gpu(sum_cpu_result, sum_gpu_result);

    // Cleanup
    CUDA_CHECK_ERROR(hipFree(d_input));
    CUDA_CHECK_ERROR(hipFree(d_sum_gpu_result));
    delete[] input;
    
    return 0;
}
